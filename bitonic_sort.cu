#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <time.h>
#include "chrono.cpp"
#include "utils.cpp"
#include <math.h>
#include <bits/stdc++.h>
 
//Every thread gets exactly one value in the unsorted array

// N_Elements Exp^2 Threads Blocks
// 1024       2^9        4     256
// 8192       2^13       8     1024
// 131072     2^17      32     4096      
// 1048576    2^21     128     16384
// 16777216   2^24     512     32768

#define THREADS 512 
#define BLOCKS 32768 
#define NUM_VALS THREADS*BLOCKS
 
struct Arranger {
    int threads, blocks, values; 
};

struct Arranger testedExecutions[5] = {
    {4, 256, 1024},
    {8, 1024, 8192},
    {32, 4096, 131072},
    {128, 16384, 1048576},
    {512, 32768, 16777216}
};

//CUDA kernel
__global__ 
void bitonic_sort_kernel(float *deviceArr, int j, int k){
    unsigned int i, ixj;
    i = threadIdx.x + blockDim.x * blockIdx.x;
    ixj = i ^ j;
 
    if ((ixj)>i) {
        if ((i & k)==0) {
            if (deviceArr[i]>deviceArr[ixj]) {
                float tmp = deviceArr[i];
                deviceArr[i] = deviceArr[ixj];
                deviceArr[ixj] = tmp;
            }
        }
        if ((i & k)!=0) {
            if (deviceArr[i] < deviceArr[ixj]) {
                float tmp = deviceArr[i];
                deviceArr[i] = deviceArr[ixj];
                deviceArr[ixj] = tmp;
            }
        }
    }
}
 
int main(int argc, char **argv){
    float *hostArrToBeSorted;
    int threadsToUse, blocksToUse = 0;
    int n_elements = 0;
    bool isPrintable = false;

    extractArgs(argc, argv, &n_elements, &isPrintable);

    if (n_elements == 0) {
        printf("Insert the size of arr after '-n' flag\n");
        return 1;
    }

    for (int i=0; i < 5; i++) {
        if (n_elements == testedExecutions[i].values) {
            threadsToUse = testedExecutions[i].threads;
            blocksToUse = testedExecutions[i].blocks;
        }
    }

    if (threadsToUse == 0 && blocksToUse == 0) {
        threadsToUse = THREADS;
        blocksToUse = BLOCKS;
    }

    chronometer_t *chrono = (chronometer_t *) malloc(sizeof(chronometer_t));

    hostArrToBeSorted = (float*) malloc(n_elements * sizeof(float));
    fillArr(hostArrToBeSorted, n_elements);

    if (isPrintable) {
        printArr(hostArrToBeSorted, n_elements);
    }
    
    float *deviceArrToBeSorted;
    hipMalloc((void**) &deviceArrToBeSorted, n_elements * sizeof(float));
    hipMemcpy(deviceArrToBeSorted, hostArrToBeSorted, n_elements * sizeof(float), hipMemcpyHostToDevice);
    
    
    dim3 blocks(blocksToUse,1); 
    dim3 threads(threadsToUse,1); 

    chrono_start(chrono);
   
    for (int k = 2; k <= blocksToUse*threadsToUse; k <<= 1) {
        for (int j= k>>1; j>0; j=j>>1) {
            bitonic_sort_kernel<<<blocks, threads>>>(deviceArrToBeSorted, j, k);
        }
    }
    
    chrono_stop(chrono);

    float *hostSortedArr = (float*) malloc(n_elements * sizeof(float));
    hipMemcpy(hostSortedArr, deviceArrToBeSorted, n_elements * sizeof(float), hipMemcpyDeviceToHost);
   
    chrono_deltaT(chrono);

    chrono_reset(chrono);

    chrono_start(chrono);
    std::sort(hostArrToBeSorted, hostArrToBeSorted+n_elements);
    chrono_stop(chrono);

    chrono_deltaT(chrono);

    if (isCorrectSolution(hostArrToBeSorted, hostSortedArr, n_elements)) {
        printf("The solution is correct!\n");
    } else {
        printf("The solution is not correct!\n");
    }

    if (isPrintable) {
        printArr(hostSortedArr, n_elements);
    }
    
    hipFree(hostArrToBeSorted);
    hipFree(deviceArrToBeSorted);
    hipFree(hostSortedArr);
    free(chrono);

    return 0;
}